#include <iostream>
#include <fstream>

#include <hipblas.h>
#include <thrust/device_vector.h>

void Prod(hipblasHandle_t handle,
          float* cdata, const float* adata, const float* bdata,
          size_t m, size_t k, size_t n) {

  float alpha = 1.0;
  float beta  = 0.0;

  size_t lda = k;
  size_t ldb = n;
  size_t ldc = n;

  hipblasOperation_t opA = HIPBLAS_OP_N;
  hipblasOperation_t opB = HIPBLAS_OP_N;

  hipblasStatus_t stat;
  stat = hipblasSgemm(handle, opB, opA, n, m, k,
                     &alpha, bdata, ldb, adata, lda, &beta, cdata, ldc);

  if(stat != HIPBLAS_STATUS_SUCCESS)
    std::abort();
}

int main(int argc, char** argv) {

  hipSetDevice(0);

  std::ifstream data("data.bin", std::ifstream::in);

  std::vector<float> hA(3072);
  data.read((char*)hA.data(), sizeof(float) * hA.size());

  std::vector<float> hB(3072 * 8);
  data.read((char*)hB.data(), sizeof(float) * hB.size());

  std::vector<float> hC(8);

  thrust::device_vector<float> dA(hA.size());
  thrust::device_vector<float> dB(hB.size());
  thrust::device_vector<float> dC(hC.size());

  thrust::copy(hA.begin(), hA.end(), dA.begin());
  thrust::copy(hB.begin(), hB.end(), dB.begin());

  hipblasHandle_t handle;
  hipblasCreate(&handle);

  const float* adata = thrust::raw_pointer_cast(dA.data());
  const float* bdata = thrust::raw_pointer_cast(dB.data());
  float* cdata = thrust::raw_pointer_cast(dC.data());

  hipblasSetMathMode(handle, HIPBLAS_DEFAULT_MATH);
  Prod(handle, cdata, adata, bdata, 1, 3072, 8);

  for(auto c : dC)
    std::cerr << c << " ";
  std::cerr << std::endl << std::endl;

  hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);
  Prod(handle, cdata, adata, bdata, 1, 3072, 8);

  for(auto c : dC)
    std::cerr << c << " ";
  std::cerr << std::endl << std::endl;

  hipblasDestroy(handle);
  return 0;
}
